
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#define MAX_SCORE_NBR 2000
#define MAX_SIT_SIZE 4000

// typedef struct bestScore BestScore;

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    char name[128];
    long size;
    Minute* minutes;
} Coin;

typedef struct {
    double score;
    long minuteId;
    long coinId;
} Score;

typedef struct {
    // POTARD
    int nbrThreads;
    int nbrBlocks;
    int sitSize;
    int nbrScores;

    Coin* source;
    Score* scores;
    Score* bestScores;
    long nbrCoins;
    Minute* src;
    int cursorCoin;
    int cursorMinute;
    Coin** coins;
} Env;

Env* e;

__global__ void compare(Env* e) {
    // int x = threadIdx.x;
    // int y = blockIdx.x;
    int workerNbr = threadIdx.x * e->nbrThreads + blockIdx.x;
    int cursorMinute = workerNbr + e->cursorMinute;
    // printf("%lf\n", e->coins[e->cursorCoin]->minutes[cursorMinute].open);
    // for (int i = 0; i < e->sitSize; i++) {
    //     Minute* minute = &e->coins[e->cursorCoin]->minutes[cursorMinute + i];

    //     // printf("%lf\n", minute->open);
    //     if (i % 100 == 0) {
    //         e->scores[workerNbr].score = i;
    //     }
    // }
    // e->scores[workerNbr].score =
    // e->coins[e->cursorCoin]->minutes[cursorMinute].volume;

    // e->scores[workerNbr].score =
    //     e->coins[e->cursorCoin]->minutes[cursorMinute].volume;

    e->scores[workerNbr].score =
        e->coins[e->cursorCoin]->minutes[cursorMinute].volume;
}

// void initBestScores() {
//     BestScore* best = e->bests;
//     best->prev = NULL;
//     for (int iBest = 1; iBest < e->nbrScores; iBest++) {
//         best->next = &e->bests[iBest];
//         best->score = NULL;
//         e->bests[iBest].prev = best;
//         best = &e->bests[iBest];
//     }
//     best->next = NULL;
//     e->lastBest = best;
// }

void printBestScores() {
    for (int i = 0; i < 20; i++) {
        printf("%lf ", e->bestScores[i].score);
    }
    printf("\n");
}

extern "C" void bake(int sitSize, Minute* minutes) {
    e->sitSize = sitSize;
    memcpy(e->src, minutes, sizeof(Minute) * sitSize);
    for (int iBest = 0; iBest < e->nbrScores; iBest++) {
        e->bestScores[iBest].score = 999999999;
    }
    for (e->cursorCoin = 0; e->cursorCoin < e->nbrCoins; e->cursorCoin++) {
        printf("%s\n", e->coins[e->cursorCoin]->name);
        e->cursorMinute = 0;
        while (1) {
            compare<<<e->nbrBlocks, e->nbrThreads>>>(e);
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(error));
                exit(-1);
            }
            // printf("%d\n", bests->score->score);
            for (int iScore = 0; iScore < e->nbrBlocks * e->nbrThreads;
                 iScore++) {
                if (e->scores[iScore].score <
                    e->bestScores[e->nbrScores - 1].score) {
                    for (int iBest = 0; iBest < e->nbrScores; iBest++) {
                        if (e->scores[iScore].score <
                            e->bestScores[iBest].score) {
                            Score tmp = e->bestScores[iBest];
                            for (int iTmp = iBest + 1; iTmp < e->nbrScores;
                                 iTmp++) {
                                Score tmp2 = e->bestScores[iTmp];
                                e->bestScores[iTmp] = tmp;
                                tmp = tmp2;
                            }
                            e->bestScores[iBest] = e->scores[iScore];
                            break;
                        }
                        // if (e->bestScores[iBest - 1].score <
                        // e->scores[iScore].score) {

                        // }
                        // if (e->scores[iScore].score >
                        //     e->bestScores[iBest].score) {
                        //         Score *tmp = e->bestScores[iBest];
                        //         e->bestScores[iBest] =
                        // }
                    }
                }
                printBestScores();
                getchar();
                // BestScore* best = e->lastBest;
                // if (((e->lastBest->score == NULL) ||
                //      e->scores[iScore].score < e->lastBest->score->score)) {
                //     while (1) {
                //         if (e->scores[iScore].score > best->score->score){
                //             e->lastBest->score = e->scores[iScore].score;
                //             e->lastBest->next
                //             e->lastBest->prev = best->prev;
                //             // best->score->next =
                //             break;
                //         }
                //         if (!best->prev) {
                //             break;
                //         }
                //         best = best->prev;
                //     }
                // }

                // for (int iBest = 0; iBest > e->nbrScores; iBest++) {
                //     if (e->scores[iScore].score < bests->score->score) {
                //         e->bests[iBest];
                //         break;
                //     }
                //     bests->score->score
                // }

                // for (int iBest = e->nbrScores - 1; iBest >= 0; iBest--) {
                //     if (e->scores[iScore].score <
                //     e->bestScores[iBest].score){
                //         e->bestScores[iBest].score = e->scores[iScore].score;
                //         e->bestScores[iBest].minuteId =
                //         e->scores[iScore].minuteId;
                //         e->bestScores[iBest].coinId =
                //         e->scores[iScore].coinId;
                //         break;
                //     }
                // }
                // printf("%.10lf\n", e->scores[iScore].score);
            }
            // exit(0);
            e->cursorMinute += e->nbrBlocks * e->nbrThreads;
            if (e->coins[e->cursorCoin]->size - e->cursorMinute <=
                e->nbrBlocks * e->nbrThreads) {
                break;
            }
        }
    }
}

extern "C" void init(int size, char* files[]) {
    hipMallocManaged(&e, sizeof(Env));
    hipMallocManaged(&e->coins, sizeof(void*) * size);
    hipMallocManaged(&e->source, sizeof(Coin));
    hipMallocManaged(&e->src, sizeof(Minute) * MAX_SIT_SIZE);
    e->nbrCoins = 0;
    e->cursorCoin = 0;
    e->nbrThreads = 256;
    e->nbrBlocks = 256;
    e->nbrScores = 100;
    hipMallocManaged(&e->scores, sizeof(Score) * e->nbrThreads * e->nbrBlocks);
    hipMallocManaged(&e->bestScores, sizeof(Score) * MAX_SCORE_NBR);
    // cudaMallocManaged(&e->bests, sizeof(Score) * MAX_SCORE_NBR);
    // e->nbrThreads = 10;
    // e->nbrBlocks = 10;

    e->sitSize = 600;
    char path[128];
    for (int i = 0; i < size; i++) {
        snprintf(path, sizeof(path), "./data/%s", files[i]);
        int fd = open(path, O_RDONLY);
        hipMallocManaged(&e->coins[i], sizeof(Coin));
        struct stat buf;
        fstat(fd, &buf);
        off_t sizeAll = buf.st_size;
        hipMallocManaged(&e->coins[i]->minutes, sizeAll);
        int res = read(fd, e->coins[i]->minutes, sizeAll);
        e->coins[i]->size = sizeAll / sizeof(Minute);
        snprintf(e->coins[i]->name, strlen(files[i]) + 1, "%s", files[i]);

        // printf("%ld -  %s\n", e->coins[i]->minutes[0].time,
        // e->coins[i]->name);

        e->nbrCoins += 1;
        close(fd);
    }
}

// int main(int argc, char* argv[]) {
//     init(argc - 1, &argv[1]);
//     bake();
//     return 0;
// }