
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#define MAX_SCORE_NBR 2000
#define MAX_SIT_SIZE 4000

// typedef struct bestScore BestScore;

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    char name[128];
    long size;
    Minute* minutes;
} Coin;

typedef struct {
    double score;
    long minuteId;
    long coinId;
} Score;

typedef struct {
    // POTARD
    int nbrThreads;
    int nbrBlocks;
    int sitSize;
    int nbrScores;

    Coin* source;
    Score* scores;
    Score* bestScores;
    long nbrCoins;
    Minute* src;
    int cursorCoin;
    int cursorMinute;
    Coin** coins;
    char* result;
} Env;

Env* e;

#define STEP_SIZE 20

__global__ void compare(Env* e) {
    int workerNbr = threadIdx.x * e->nbrThreads + blockIdx.x;
    int cursorMinute = workerNbr + e->cursorMinute;
    double score = 0;
    int step = 0;
    for (int i = 1; i < e->sitSize; i++) {
        double destPourcent =
            e->coins[e->cursorCoin]->minutes[cursorMinute + step].open /
            e->coins[e->cursorCoin]->minutes[cursorMinute + i].open * 1000;
        double srcPourcent = e->src[0 + step].open / e->src[i].open * 1000;
        score += abs(destPourcent - srcPourcent);
        if (step > STEP_SIZE){
            step += 1;
        }
            
    }
    e->scores[workerNbr].score = score;
    e->scores[workerNbr].minuteId = cursorMinute;
    e->scores[workerNbr].coinId = e->cursorCoin;
}

void printBestScores() {
    for (int i = 0; i < e->nbrScores; i++) {
        // for (int i = 0; i < 2; i++) {
        printf("%.15lf %s %ld\n", e->bestScores[i].score,
               e->coins[e->bestScores[i].coinId]->name,
               e->coins[e->bestScores[i].coinId]
                   ->minutes[e->bestScores[i].minuteId]
                   .time);
    }
    printf("\n");
}

extern "C" char* bake(int sitSize, Minute* minutes) {
    e->sitSize = sitSize;
    memcpy(e->src, minutes, sizeof(Minute) * sitSize);
    for (int iBest = 0; iBest < e->nbrScores; iBest++) {
        e->bestScores[iBest].score = 999999999999;
    }
    for (e->cursorCoin = 0; e->cursorCoin < e->nbrCoins; e->cursorCoin++) {
        // printf("%s\n", e->coins[e->cursorCoin]->name);
        e->cursorMinute = 0;
        while (1) {
            compare<<<e->nbrBlocks, e->nbrThreads>>>(e);
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(error));
                exit(-1);
            }
            for (int iScore = 0; iScore < e->nbrBlocks * e->nbrThreads;
                 iScore++) {
                if (e->scores[iScore].score <=
                    e->bestScores[e->nbrScores - 1].score) {
                    // printf("%lf %lf %s\n",
                    // e->scores[iScore].score,e->coins[e->scores[iScore].coinId]->minutes[e->scores[iScore].minuteId].volume,
                    // e->coins[e->scores[iScore].coinId]->name);

                    for (int iBest = 0; iBest < e->nbrScores; iBest++) {
                        if (e->scores[iScore].score <
                            e->bestScores[iBest].score) {
                            Score tmp = e->bestScores[iBest];
                            for (int iTmp = iBest + 1; iTmp < e->nbrScores;
                                 iTmp++) {
                                Score tmp2 = e->bestScores[iTmp];
                                e->bestScores[iTmp] = tmp;
                                tmp = tmp2;
                            }
                            e->bestScores[iBest] = e->scores[iScore];
                            break;
                        }
                    }
                }
            }
            // exit(0);
            e->cursorMinute += e->nbrBlocks * e->nbrThreads;
            if (e->coins[e->cursorCoin]->size - e->cursorMinute <=
                e->nbrBlocks * e->nbrThreads) {
                break;
            }
        }
    }
    // printBestScores();
    int nbrChars = 0;
    for (int i = 0; i < e->nbrScores; i++) {
        nbrChars += sprintf(
            &e->result[nbrChars], "%lf|%s|%ld\n", e->bestScores[i].score,
            e->coins[e->bestScores[i].coinId]->name, e->bestScores[i].minuteId);
    }
    return e->result;
    // printf("%s\n", e->result);
}

extern "C" void init(int size, char* files[]) {
    hipMallocManaged(&e, sizeof(Env));
    hipMallocManaged(&e->coins, sizeof(void*) * size);
    hipMallocManaged(&e->source, sizeof(Coin));
    hipMallocManaged(&e->src, sizeof(Minute) * MAX_SIT_SIZE);
    e->result = (char*)malloc(MAX_SCORE_NBR * 1024);

    e->nbrCoins = 0;
    e->cursorCoin = 0;
    e->nbrBlocks = 256;
    e->nbrThreads = 128;
    e->nbrScores = 2000;

    e->nbrBlocks = 256;
    e->nbrThreads = 256;

    hipMallocManaged(&e->scores, sizeof(Score) * e->nbrThreads * e->nbrBlocks);
    hipMallocManaged(&e->bestScores, sizeof(Score) * MAX_SCORE_NBR);
    e->sitSize = 600;
    char path[128];
    for (int i = 0; i < size; i++) {
        snprintf(path, sizeof(path), "./data/%s", files[i]);
        int fd = open(path, O_RDONLY);
        if (fd < 0) {
            continue;
        }
        hipMallocManaged(&e->coins[i], sizeof(Coin));
        struct stat buf;
        fstat(fd, &buf);
        off_t sizeAll = buf.st_size;
        hipMallocManaged(&e->coins[i]->minutes, sizeAll);
        int res = read(fd, e->coins[i]->minutes, sizeAll);
        e->coins[i]->size = sizeAll / sizeof(Minute);
        snprintf(e->coins[i]->name, strlen(files[i]) + 1, "%s", files[i]);
        e->nbrCoins += 1;
        close(fd);
    }
}