#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#define MAX_SCORE_NBR 10000
#define MAX_SIT_SIZE 2000

typedef struct bestScore BestScores;

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    char name[128];
    long size;
    Minute* minutes;
} Coin;

typedef struct {
    double score;
    long minuteId;
    long coinId;
} Score;

typedef struct bestScore{
    Score *score;
    BestScores *next;
};

typedef struct {
    // POTARD
    int nbrThreads;
    int nbrBlocks;
    int sitSize;
    int nbrScores;

    Coin* source;
    Score* scores;
    long nbrCoins;
    Minute* src;
    int cursorCoin;
    int cursorMinute;
    Coin** coins;
} Env;

Env* e;

__global__ void compare(Env* e) {
    // int x = threadIdx.x;
    // int y = blockIdx.x;
    int workerNbr = threadIdx.x * e->nbrThreads + blockIdx.x;
    int cursorMinute = workerNbr + e->cursorMinute;
    // printf("%lf\n", e->coins[e->cursorCoin]->minutes[cursorMinute].open);
    // for (int i = 0; i < e->sitSize; i++) {
    //     Minute* minute = &e->coins[e->cursorCoin]->minutes[cursorMinute + i];

    //     // printf("%lf\n", minute->open);
    //     if (i % 100 == 0) {
    //         e->scores[workerNbr].score = i;
    //     }
    // }
    // e->scores[workerNbr].score =
    // e->coins[e->cursorCoin]->minutes[cursorMinute].volume;

    // e->scores[workerNbr].score =
    //     e->coins[e->cursorCoin]->minutes[cursorMinute].volume;

    e->scores[workerNbr].score =
        e->coins[e->cursorCoin]->minutes[cursorMinute].volume;

}

extern "C" void bake(int sitSize, Minute* minutes) {
    e->sitSize = sitSize;
    memcpy(e->src, minutes, sizeof(Minute) * sitSize);
    for (int i = 0; i < e->nbrScores; i++) {
        e->bestScores[i].score = 999999999999;
        e->bestScores[i].minuteId = 0;
        e->bestScores[i].coinId = 0;
    }
    for (e->cursorCoin = 0; e->cursorCoin < e->nbrCoins; e->cursorCoin++) {
        printf("%s\n", e->coins[e->cursorCoin]->name);
        e->cursorMinute = 0;
        while (1) {
            compare<<<e->nbrBlocks, e->nbrThreads>>>(e);
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(error));
                exit(-1);
            }

            for (int iScore = 0; iScore < e->nbrBlocks * e->nbrThreads;
                 iScore++) {
                for (int iBest = e->nbrScores - 1; iBest >= 0; iBest--) {
                    if (e->scores[iScore].score <
                    e->bestScores[iBest].score){
                        e->bestScores[iBest].score = e->scores[iScore].score;
                        e->bestScores[iBest].minuteId =
                        e->scores[iScore].minuteId;
                        e->bestScores[iBest].coinId =
                        e->scores[iScore].coinId; 
                        break;
                    }
                }
                // printf("%.10lf\n", e->scores[iScore].score);
            }

            // exit(0);
            e->cursorMinute += e->nbrBlocks * e->nbrThreads;
            if (e->coins[e->cursorCoin]->size - e->cursorMinute <=
                e->nbrBlocks * e->nbrThreads) {
                break;
            }
        }
    }
}

extern "C" void init(int size, char* files[]) {
    hipMallocManaged(&e, sizeof(Env));
    hipMallocManaged(&e->coins, sizeof(void*) * size);
    hipMallocManaged(&e->source, sizeof(Coin));
    hipMallocManaged(&e->src, sizeof(Minute) * MAX_SIT_SIZE);
    e->nbrCoins = 0;
    e->cursorCoin = 0;
    e->nbrThreads = 256;
    e->nbrBlocks = 256;
    e->nbrScores = 20;
    hipMallocManaged(&e->scores, sizeof(Score) * e->nbrThreads * e->nbrBlocks);
    hipMallocManaged(&e->bestScores, sizeof(Score) * MAX_SCORE_NBR);
    // e->nbrThreads = 10;
    // e->nbrBlocks = 10;

    e->sitSize = 600;
    char path[128];
    for (int i = 0; i < size; i++) {
        snprintf(path, sizeof(path), "./data/%s", files[i]);
        int fd = open(path, O_RDONLY);
        hipMallocManaged(&e->coins[i], sizeof(Coin));
        struct stat buf;
        fstat(fd, &buf);
        off_t sizeAll = buf.st_size;
        hipMallocManaged(&e->coins[i]->minutes, sizeAll);
        int res = read(fd, e->coins[i]->minutes, sizeAll);
        e->coins[i]->size = sizeAll / sizeof(Minute);
        snprintf(e->coins[i]->name, strlen(files[i]) + 1, "%s", files[i]);

        // printf("%ld -  %s\n", e->coins[i]->minutes[0].time,
        // e->coins[i]->name);

        e->nbrCoins += 1;
        close(fd);
    }
}

// int main(int argc, char* argv[]) {
//     init(argc - 1, &argv[1]);
//     bake();
//     return 0;
// }