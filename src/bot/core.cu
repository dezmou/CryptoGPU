
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    char name[128];
    long size;
    Minute* minutes;
} Coin;

typedef struct {
    int score;
    int minuteId;
    int coinId;
} Score;

typedef struct {
    // POTARD
    int nbrThreads;
    int nbrBlocks;
    int sitSize;
    
    Coin *source;
    long nbrCoins;
    int cursorCoin;
    int cursorMinute;
    Coin** coins;
} Env;

Env* e;

__global__ void compare(Env* e) {
    // int x = threadIdx.x;
    // int y = blockIdx.x;
    int cursorMinute = threadIdx.x * e->nbrThreads + blockIdx.x + e->cursorMinute;
}

void init(int size, char* files[]) {
    hipMallocManaged(&e->coins, sizeof(void*) * size);
    hipMallocManaged(&e->source, sizeof(Coin));
    e->nbrCoins = 0;
    e->cursorCoin = 0;
    e->nbrThreads = 128;
    e->nbrBlocks = 128;

    // e->nbrThreads = 10;
    // e->nbrBlocks = 10;

    e->sitSize = 400;
    char path[128];
    for (int i = 0; i < size; i++) {
        snprintf(path, sizeof(path), "./data/%s", files[i]);
        int fd = open(path, O_RDONLY);
        hipMallocManaged(&e->coins[i], sizeof(Coin));
        struct stat buf;
        fstat(fd, &buf);
        off_t sizeAll = buf.st_size;
        hipMallocManaged(&e->coins[i]->minutes, sizeAll);
        int res = read(fd, e->coins[i]->minutes, sizeAll);
        e->coins[i]->size = sizeAll / sizeof(Minute);
        snprintf(e->coins[i]->name, strlen(files[i]), "%s", files[i]);
        // printf("%ld %s\n", e->coins[i]->minutes[0].time, e->coins[i]->name);
        e->nbrCoins += 1;
        close(fd);
    }
}

void bake() {
    // cudaDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    for (e->cursorCoin; e->cursorCoin < e->nbrCoins; e->cursorCoin++) {
        printf("%s\n", e->coins[e->cursorCoin]->name);
        e->cursorMinute = 0;
        while (1) {
            e->cursorMinute += e->nbrBlocks * e->nbrThreads;
            compare<<<e->nbrBlocks, e->nbrThreads>>>(e);
            hipDeviceSynchronize();
            if (e->cursorMinute >= e->coins[e->cursorCoin]->size - e->sitSize){
                break;
            }
        }
    }
}

int main(int argc, char* argv[]) {
    hipMallocManaged(&e, sizeof(Env));
    init(argc - 1, &argv[1]);
    bake();
    return 0;
}