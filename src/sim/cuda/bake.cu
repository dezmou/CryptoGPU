#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

#include "trade.h"

#define TIME_START 50
#define AMOUNT_STOP 9999999999999
#define FEE_TAKER 0.0004
#define FEE_MAKER 0.0004
#define BET_AMOUNT 10000

typedef struct {
    Data data;
    int nbrThreads;
    int nbrBlocks;
    Potards *potards;
    Broker *brokers;
} BakeParams;

__device__ double getVariance(Minute *minute, Potards *potards) {
    double min = 99999999999;
    double max = -9999999999;
    for (int cursor = -(potards->period_for_variance + 1); cursor <= 0;
         cursor++) {
        if (minute[cursor].low < min) {
            min = minute[cursor].low;
        }
        if (minute[cursor].high > max) {
            max = minute[cursor].high;
        }
    }
    return (max / min * 100) - 100;
}

__device__ Bet newBet(Minute *minute, int type, double amount, double closeWin,
           double closeLose) {
    Bet bet;
    bet.type = type;
    bet.totalFee = 0;
    bet.amount = amount;
    bet.closeLose = 0;
    bet.closeWin = 0;
    if (bet.type == NO_BET) {
        return bet;
    } else if (bet.type == SELL) {
        bet.closeLose = minute->close * (1 + closeLose * 0.01);
        bet.closeWin = minute->close * (1 - closeWin * 0.01);
    } else if (bet.type == BUY) {
        bet.closeLose = minute->close * (1 - closeLose * 0.01);
        bet.closeWin = minute->close * (1 + closeWin * 0.01);
    }
    return bet;
}

__device__ Bet analyse(Minute *minute, Potards *potards) {
    // dev
    // return newBet(minute, SELL, BET_AMOUNT / minute->close,
    //               potards->closeWin ,
    //               potards->closeLose);

    double change_before_long =
        100 - (minute[-(potards->change_before_long_steps)].close /
               minute->close * 100);
    // double variance = getVariance(minute, potards);
    double variance = 5.0;
    if (change_before_long > potards->change_before_long &&
        variance < potards->maxVariance) {
        return newBet(minute, SELL, BET_AMOUNT / minute->close,
                      potards->closeWin * change_before_long,
                      potards->closeLose * change_before_long);
    } else {
        return newBet(NULL, NO_BET, 0, 0, 0);
    }
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

__device__ void openBet(Broker *broker, Bet *bet) {
    double usd = bet->amount * broker->minutes[broker->cursor].close;
    double fee = usd * FEE_MAKER;
    bet->price = broker->minutes[broker->cursor].close;
    broker->bank += -usd;
    broker->bank += -fee;
    broker->bet = bet;
    broker->totalFee += fee;
    bet->startCursor = broker->cursor;
    // if (!LEARN) {
    //     printMinute(&broker->minutes[broker->cursor]);
    // }
}

__device__ void closeBet(Broker *broker, Bet *bet, double price) {
    double usd;
    double closePrice = price;
    if (bet->type == BUY) {
        // printf("CHENAPAN\n");
        usd = bet->amount * closePrice;
    } else if (bet->type == SELL) {
        usd = bet->amount * (bet->price + (bet->price - closePrice));
    }
    double fee = usd * FEE_MAKER;
    broker->bank += usd;
    broker->bank += -fee;
    broker->bet = NULL;
    broker->totalFee += fee;
    broker->nbrBets += 1;
    // printf("chien\n");
    // if (!LEARN) {
    //     printMinute(&broker->minutes[broker->cursor]);
    //     fprintf(fp, "%lf,%lf,%lf\n", broker->minutes[broker->cursor].close,
    //             broker->bank, broker->totalFee);
        // printf(
        //     "%-4s STA %-7ld CLO: %-7ld BK: %-9.2lf PRI: %-10.2lf CLW:
        //     %-10.2lf " "CLL: "
        //     "%-10.2lf "
        //     "ACT: %-10.2lf FEE: %-10.5lf TFE: %-10.5lf  NBT: %-6d FL: "
        //     "%-5d\n\n\n",
        //     (bet->type == BUY ? "BUY" : "SELL"), bet->startCursor + 2,
        //     broker->cursor + 2, broker->bank, bet->price, bet->closeWin,
        //     bet->closeLose, broker->minutes[broker->cursor].close, fee,
        //     broker->totalFee, broker->nbrBets, broker->flatScore);
    // }
    // printf(KWHT);
    // getchar();
}

__device__ int tickBroker(Broker *broker) {
    do {
        broker->cursor += 1;
        // if (broker->cursor % 100000 == 0){
        //     if (broker->bank < 0){
        //         return 0;
        //     }
        // }

        if (broker->cursor % 50000 == 0) {
            if (broker->bank > broker->lastFlatBank) {
                broker->flatScore += 1;
            } else if (broker->bank < broker->lastFlatBank) {
                broker->flatScore += -1;
            }
            // broker->flatScore += (broker->bank > broker->lastFlatBank) ? 1 :
            // 0;
            broker->lastFlatBank = broker->bank;
            broker->nbrFlatScore += 1;
        }
        if (broker->bet) {
            Bet *bet = broker->bet;
            if (bet->type == BUY) {
                if (broker->minutes[broker->cursor].low <= bet->closeLose) {
                    // lose
                    // printf(KRED);
                    closeBet(broker, bet, bet->closeLose);
                    broker->nbrLost++;
                } else if (broker->minutes[broker->cursor].high >=
                           bet->closeWin) {
                    // WIN
                    // printf(KGRN);
                    closeBet(broker, bet, bet->closeWin);
                    broker->nbrWon++;
                }
            } else if (bet->type == SELL) {
                if (broker->minutes[broker->cursor].high >= bet->closeLose) {
                    // lose
                    // printf(KRED);
                    closeBet(broker, bet, bet->closeLose);
                    broker->nbrLost++;
                } else if (broker->minutes[broker->cursor].low <=
                           bet->closeWin) {
                    // WIN
                    // printf(KGRN);
                    closeBet(broker, bet, bet->closeWin);
                    broker->nbrWon++;
                }
            }
        }
        if (broker->cursor >= broker->nbrMinutes - 1 ||
            (broker->cursor - TIME_START) > AMOUNT_STOP) {
            return 0;
        }
    } while (broker->bet != NULL);
    return 1;
}

__device__ void bake(Potards *potards, Broker *broker) {
    // printf("%lf\n", broker->minutes->open);
    do {
        Bet bet = analyse(&broker->minutes[broker->cursor], potards);
        if (bet.type) {
            openBet(broker, &bet);
        }
    } while (tickBroker(broker));
}

__device__ void printMinute(Minute *minute) {
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}


__device__ Broker newBroker(Data data) {
    // Broker *broker = malloc(sizeof(Broker));
    Broker broker;
    broker.cursor = TIME_START;
    broker.bet = NULL;
    broker.minutes = data.minutes;
    broker.bank = 0;
    broker.nbrBets = 0;
    broker.totalFee = 0;
    broker.nbrWon = 0;
    broker.nbrLost = 0;
    broker.flatScore = 0;
    broker.nbrFlatScore = 0;
    broker.lastFlatBank = 0;
    broker.variance = 0;
    broker.nbrMinutes = data.nbrMinutes;
    return broker;
}

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}


Potards newPotards() {
    Potards res;
    res.change_before_long = 1.35;
    res.change_before_long_steps = 1;
    res.closeWin = 0.64;
    res.closeLose = 3.55;
    res.period_for_variance = 40;
    res.maxVariance = 17.11;
    // if (LEARN) {
    
    res.change_before_long = randfrom(0.01, 0.8);
    // res.change_before_long_steps = (long)randfrom(1, 10);
    res.closeWin = randfrom(0.05, 1);
    res.closeLose = randfrom(0.05, 1);
    // res.maxVariance = randfrom(1, 20);
    
    // }
    return res;
}

__global__ void cudaBake(BakeParams *p) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x; 
    Potards potard = p->potards[workerNbr];
    Broker broker = newBroker(p->data);
    bake(&potard, &broker);
    p->brokers[workerNbr] = broker;
    // p->brokers[workerNbr] = broker;
    printf("THX: %-5d BLX: %-5d THY: %-5d BLY: %-5d\n", threadIdx.x, blockIdx.x, threadIdx.y, blockIdx.y);
    // printMinute(&p->data.minutes[workerNbr]);
}

#define NBR_THREAD 512
#define NBR_BLOCK 512

// #define NBR_THREAD 1
// #define NBR_BLOCK 1


int main() {
    // return 0;
    BakeParams *p;
    hipMallocManaged(&p, sizeof(BakeParams));
    hipMallocManaged(&p->potards, sizeof(Potards) * NBR_BLOCK * NBR_THREAD);
    hipMallocManaged(&p->brokers, sizeof(Broker) * NBR_BLOCK * NBR_THREAD);
    p->nbrBlocks = NBR_BLOCK;
    p->nbrThreads = NBR_THREAD;
    p->data = loadMinutes("../../../data/bin/BTCUSDT");

    for (int i = 0; i < NBR_THREAD * NBR_BLOCK; i++) {
        p->potards[i] = newPotards();
        // p->brokers[i] = newBroker(p->data);
    }
    cudaBake<<<p->nbrBlocks, p->nbrThreads>>>(p);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    return 0;
    for (int i = 0; i < NBR_THREAD * NBR_BLOCK; i++) {
        printf(
            "BK: %-8.2lf  NB: %-5d CBL: %-8.2lf CBLS: %-8.2ld CLW: %-8.2lf "
            "CLS: %-8.2lf FEE: %-8.2lf  NBW: %5ld NBL: %5ld "
            "FL: %-5d MXV: %-8.2lf\n",
            p->brokers[i].bank, p->brokers[i].nbrBets, p->potards[i].change_before_long,
            p->potards[i].change_before_long_steps, p->potards[i].closeWin, p->potards[i].closeLose,
            p->brokers[i].totalFee, p->brokers[i].nbrWon, p->brokers[i].nbrLost,
            p->brokers[i].flatScore, p->potards[i].maxVariance);
    }

    return 0;
}
