#include "hip/hip_runtime.h"
#include "trade.h"

#define FEE_TAKER 0.0004
#define FEE_MAKER 0.0002

Broker newBroker(Data data) {
    Broker broker;
    broker.cursor = 0;
    broker.minutes = data.minutes;
    broker.nbrMinutes = data.nbrMinutes;
    broker.bank = 0;
    broker.seed = plantSeed();
    broker.fees = 0;
    broker.bet.type = NO_BET;
    broker.bet.closeDown = 0;
    broker.bet.closeUp = 0;
    broker.nbrBets = 0;
    broker.reg = 0;
    broker.lastRegBank = broker.bank;
    return broker;
}

#define MINUTE broker->minutes[broker->cursor]
#define SIZE_BET 4660

__host__
#ifndef PLAY
    __device__
#endif
    static void
    closeBet(Broker *broker, int isWin, double diff) {
    broker->bet.totalFee += SIZE_BET * FEE_TAKER;
    double gain = SIZE_BET * (isWin == 1 ? diff : -diff);
    broker->bank += gain;
    broker->bet.totalFee = (FEE_TAKER * SIZE_BET);
    broker->bet.totalFee +=
        ((isWin == 1 ? FEE_TAKER : FEE_MAKER) * (SIZE_BET + gain));
    broker->fees += broker->bet.totalFee;
    broker->bank += -broker->bet.totalFee;
    broker->nbrBets += 1;

#ifdef PLAY
    fprintf(fp, "%lf,%lf,%lf\n", broker->minutes[broker->cursor].close,
            broker->bank, broker->fees);

    printf(
        "%s%-4s DIFF: %-5.04lf STH: %-5.04lf STL: %-5.04lf GAIN: "
        "%-5.04lf FEE :%-8.05lf\n",
        (isWin == 1 ? "\x1B[32m" : "\x1B[31m"),
        (broker->bet.type == SELL ? "SELL" : "BUY"), diff, broker->bet.closeUp,
        broker->bet.closeDown, gain, broker->bet.totalFee);
    printMinute(&broker->minutes[broker->bet.cursor], broker->bet.cursor);
    printMinute(&broker->minutes[broker->cursor], broker->cursor);
    printf("BK: %-8.04lf FEE: %-8.02lf NB: %-5d\n", broker->bank, broker->fees,
           broker->nbrBets);
    printf(
        "\x1B[0m---------------------------------------------------------------"
        "-------------------------------------------\n");
#endif
    broker->bet.type = NO_BET;
}

__host__ __device__ void tickBroker(Broker *broker) {
    if (broker->cursor % BROKER_REG_STEP == 0){
        broker->reg += (broker->bank > broker->lastRegBank) ? 1 : -1;
        broker->lastRegBank = broker->bank;
    }
    if (broker->bet.type == NO_BET) {
        broker->bet = analyse(&MINUTE, &broker->seed);
        if (broker->bet.type != NO_BET) {
            broker->bet.cursor = broker->cursor;
        }
        return;
    } else if (broker->bet.type == SELL) {
        if (MINUTE.high >= broker->bet.closeUp) {
            double diff = fabs((broker->bet.closeUp /
                                broker->minutes[broker->bet.cursor].close) -
                               1);
            // LOSE
            closeBet(broker, 0, diff);
        } else if (MINUTE.low < broker->bet.closeDown) {
            // WIN
            double diff = fabs((broker->bet.closeDown /
                                broker->minutes[broker->bet.cursor].close) -
                               1);
            closeBet(broker, 1, diff);
        }
    } else if (broker->bet.type == BUY) {
        if (MINUTE.low <= broker->bet.closeDown) {
            // LOSE
            double diff = fabs((broker->bet.closeDown /
                                broker->minutes[broker->bet.cursor].close) -
                               1);
            closeBet(broker, 0, diff);
        } else if (MINUTE.high > broker->bet.closeUp) {
            // WIN
            double diff = fabs((broker->bet.closeUp /
                                broker->minutes[broker->bet.cursor].close) -
                               1);
            closeBet(broker, 1, diff);
        }
    }
}
