#include "hip/hip_runtime.h"
#include "trade.h"

Seed plantSeed() {
    Seed seedRes;
    Seed *seed = &seedRes;
#define S_CURSOR_BACK seed->a
    S_CURSOR_BACK = randfrom(300, 10);
#define S_BACK_MAX seed->b
    S_BACK_MAX = randfrom(5, 50);
#define S_BACK_MIN seed->c
    S_BACK_MIN = randfrom(-5, -50);
#define S_CHANGE_ACT_MIN seed->d
    S_CHANGE_ACT_MIN = randfrom(-0.5, 0.5);
#define S_CHANGE_LAST_MIN seed->e
    S_CHANGE_LAST_MIN = randfrom(0.1, 4);
#define S_STOP_HIGH seed->f
    S_STOP_HIGH = randfrom(0.001, 0.2);
#define S_STOP_LOW seed->f
    S_STOP_LOW = randfrom(0.001, 0.2);

    // S_BACK_MAX = 20;
    // S_BACK_MIN = -10;
    // S_CURSOR_BACK = 400;
    // S_CHANGE_ACT_MIN = 2;
    // S_CHANGE_LAST_MIN = 3;
    return seedRes;
}

DEVICE void static initBet(Bet *bet, int type, double closeUp,
                           double closeDown) {
    bet->type = type;
    bet->closeUp = closeUp;
    bet->closeDown = closeDown;
}

DEVICE void analyse(Minute *minute, Seed *seed, Bet *bet) {
    bet->type = NO_BET;
    double allChange = 0;
    double allChangeNbr = 0;
    double backMin = 99999999;
    double backMax = -99999999;
    for (int i = -S_CURSOR_BACK; i < -6; i+=5) {
        allChange += fabs(100 - (minute[i].open / minute[i].close * 100));
        allChangeNbr += 1;
        if (minute[i].close > backMax) {
            backMax = minute[i].close;
        }
        if (minute[i].close < backMin) {
            backMin = minute[i].close;
        }
    }
    double variance = allChange / allChangeNbr;
    backMax = (100 - (minute->close / backMax * 100)) / variance;
    backMin = (100 - (minute->close / backMin * 100)) / variance;
    if (backMax < S_BACK_MAX && backMin > S_BACK_MIN) {
        double changeAct = 100 - (minute->open / minute->close * 100);
        double changeLast = 100 - (minute[-1].open / minute[0].close * 100);
        double changeActVar = changeAct / variance;
        double changeLastVar = changeLast / variance;
        if (changeLastVar > S_CHANGE_LAST_MIN && changeAct > 0) {
            // printf("VA: %-8.3lf BACKMAX: %-8.3lf BACKMIN: %-8.3lf\n", variance,
            //        backMax, backMin);
            initBet(bet, SELL, minute->close + (minute->close * variance * S_STOP_HIGH), minute->close - (minute->close * variance * S_STOP_LOW));
        }
    }
}
