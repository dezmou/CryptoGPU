#include "hip/hip_runtime.h"
#include "trade.h"

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

void printSeed(Seed *seed) {
    printf("CH:%-12.02lf CH:%-12.02lf\n", seed->chien, seed->lapin);
}

Seed scanSeed(char *seedStr) {
    Seed seed;
    sscanf(seedStr, "CH:%lf LP:%lf\n", &seed.chien, &seed.lapin);
    return seed;
}

Seed plantSeed() {
    Seed seed;
    seed.chien = randfrom(5, 14000);
    return seed;
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
#ifdef PLAY
    data.minutes = (Minute*)malloc(size);
#endif
#ifndef PLAY
    hipMallocManaged(&data.minutes, size);
#endif
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

__host__ __device__ void printMinute(Minute *minute, int cursor) {
    if (cursor != -1) {
        printf("~> %-6d | ", cursor + 2);
    }
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}