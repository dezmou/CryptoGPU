#include "hip/hip_runtime.h"
#include "trade.h"

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

#define SEEDSTR \
    "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf\n"
#define SEEDPARAMS

void printSeed(Seed *seed) {
    printf(SEEDSTR, seed->a, seed->b, seed->c, seed->d, seed->e, seed->f,
           seed->g, seed->h, seed->i, seed->j, seed->k, seed->l, seed->m,
           seed->n, seed->o, seed->p);
}

Seed scanSeed(char *seedStr) {
    Seed seed;
    sscanf(seedStr, SEEDSTR, &seed.a, &seed.b, &seed.c, &seed.d, &seed.e,
           &seed.f, &seed.g, &seed.h, &seed.i, &seed.j, &seed.k, &seed.l,
           &seed.m, &seed.n, &seed.o, &seed.p);
    return seed;
}

DEVICE void printMinute2(Line *line, int cursor) {
    if (cursor != -1) {
        printf("~> %-6d | ", cursor + 2);
    }
    printf(
        "%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf "
        "AVG_C: %-10.5lf\n",
        line->time, line->open, line->high, line->low, line->close,
        line->avgCandle);
}

void createIndicators(Data *data) {
#ifdef PLAY
    data->line = (Line *)malloc(sizeof(Line) * data->nbrMinutes);
#endif
#ifndef PLAY
    hipMallocManaged(&data->line, sizeof(Line) * data->nbrMinutes);
#endif
    for (int i = 0; i < data->nbrMinutes; i++) {
        double avg = -1;

        if (i > 1405) {
            double totalCandleSize = 0;
            int nbrCandles = 0;
            for (int j = i - 150; j < i - 10; j++) {
                // &data->minutes[j];
                nbrCandles += 1;
                totalCandleSize +=
                    fabs((data->minutes[j].open - data->minutes[j].close));
            }
            avg = totalCandleSize / nbrCandles;
        }
        memcpy(&data->line[i], &data->minutes[i], sizeof(Minute));
        data->line[i].avgCandle = avg;
        printMinute2(&data->line[i], i);
    }
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
#ifdef PLAY
    data.minutes = (Minute *)malloc(size);
#endif
#ifndef PLAY
    hipMallocManaged(&data.minutes, size);
#endif
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    createIndicators(&data);
    return data;
}

DEVICE void printMinute(Minute *minute, int cursor) {
    if (cursor != -1) {
        printf("~> %-6d | ", cursor + 2);
    }
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}