#include "hip/hip_runtime.h"
#include "trade.h"

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

#define SEEDSTR "%lf %lf %lf\n"
#define SEEDPARAMS

void printSeed(Seed *seed) {
    printf(SEEDSTR, seed->a, seed->b, seed->c);
}

Seed scanSeed(char *seedStr) {
    Seed seed;
    sscanf(seedStr, SEEDSTR, &seed.a, &seed.b, &seed.c);
    return seed;
}


Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
#ifdef PLAY
    data.minutes = (Minute*)malloc(size);
#endif
#ifndef PLAY
    hipMallocManaged(&data.minutes, size);
#endif
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

__host__ __device__ void printMinute(Minute *minute, int cursor) {
    if (cursor != -1) {
        printf("~> %-6d | ", cursor + 2);
    }
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}