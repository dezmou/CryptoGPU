#include "hip/hip_runtime.h"
#include "trade.h"

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

void printSeed(Seed *seed) { 
    printf("CH:%-12.02lf\n", seed->chien); 
}

Seed scanSeed(char *seedStr){
    Seed seed;
    sscanf(seedStr,"CH:%lf\n", &seed.chien);
    return seed;
}

Seed plantSeed() {
    Seed seed;
    seed.chien = randfrom(5, 14000);
    return seed;
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

__host__ __device__ void printMinute(Minute *minute) {
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}