#include "hip/hip_runtime.h"
#include "trade.h"

double randfrom(double min, double max) {
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

__host__ __device__ void printMinute(Minute *minute) {
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close);
}

__global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;

    Broker broker = brokers[workerNbr];
    broker.cursor = cursor;

    // printf("%d\n", broker.cursor);
    if (broker.minutes[broker.cursor].open > broker.seed.chien) {
        broker.bank += 1;
    }

    brokers[workerNbr] = broker;
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

Seed plantSeed() {
    Seed seed;
    seed.chien = randfrom(5, 14000);
    return seed;
}

Broker newBroker(Data data) {
    // Broker *broker = malloc(sizeof(Broker));
    Broker broker;
    broker.cursor = 700000;
    broker.minutes = data.minutes;
    broker.nbrMinutes = data.nbrMinutes;
    broker.seed = plantSeed();
    broker.bank = 0;
    return broker;
}

// #define TIME_START 700000
#define TIME_START 0

void bake(Data data) {
    int nbrThreads = 128;
    int nbrBlocks = 128;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    for (int i = 0; i < nbrWorkers; i++) {
        brokers[i] = newBroker(data);
    }
    for (int i = TIME_START; i < data.nbrMinutes; i++) {
        if (i % 1000 == 0) {
            printf("%d / 800000\n", i);
        }
        applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
    }
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    for (int i = 0; i < nbrWorkers; i++) {
        printf("CH: %-12.2lf BK: %-12.2lf\n", brokers[i].seed.chien,
               brokers[i].bank);
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"../../data/bin/BTCUSDT");
    bake(data);
    return 0;
}