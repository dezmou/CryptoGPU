#include "hip/hip_runtime.h"
#include "trade.h"

static __global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    Broker broker = brokers[workerNbr];
    broker.cursor = cursor;
    tickBroker(&broker);
    brokers[workerNbr] = broker;
}

// #define TIME_START 700000
#define TIME_START 0

static void bake(Data data) {
    int nbrThreads = 128;
    int nbrBlocks = 128;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    for (int i = 0; i < nbrWorkers; i++) {
        brokers[i] = newBroker(data);
    }
    for (int i = TIME_START; i < data.nbrMinutes; i++) {
        if (i % 1000 == 0) {
            printf("%d / 800000  wokers : %d\n", i, nbrWorkers);
        }
        applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
    }

    
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    for (int i = 0; i < nbrWorkers; i++) {
        printf("BK: %-8.02lf FEE: %-8.02lf NB: %-5d\n", brokers[i].bank, brokers[i].fees, brokers[i].nbrBets);
        // printf("BK: %-12.2lf\n\n", brokers[i].bank);
        printSeed(&brokers[i].seed);
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"../../data/bin/BTCUSDT");
    bake(data);
    return 0;
}