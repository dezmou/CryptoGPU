#include "hip/hip_runtime.h"
#include "trade.h"

static __global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    brokers[workerNbr].cursor = cursor;
    tickBroker(&brokers[workerNbr]);
}

// #define TIME_START 700000
#define TIME_START 500

static void bake(Data data) {
    int nbrThreads = 64;
    int nbrBlocks = 127;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    double maxBank = -999999999;
    double maxReg = 10;
 
    for (int chien = 0; chien < 100000; chien++) {
        for (int i = 0; i < nbrWorkers; i++) {
            brokers[i] = newBroker(data);
        }
        for (int i = TIME_START; i < data.nbrMinutes; i++) {
            if (i % 100000 == 0) {
                printf("%d / 800000  wokers : %d\n", i, nbrWorkers);
            }
            applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
        }
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < nbrWorkers; i++) {
            // printf("BK: %-12.2lf\n\n", brokers[i].bank);
            if (brokers[i].bank > maxBank && brokers[i].reg >= maxReg && brokers[i].nbrBets > BAKE_MIN_BETS) {
                printSeed(&brokers[i].seed);
                printf("BK: %-8.02lf FEE: %-8.02lf NB: %-5d REG: %-5d\n\n",
                       brokers[i].bank, brokers[i].fees, brokers[i].nbrBets, brokers[i].reg);
                maxBank = brokers[i].bank;
                maxReg = brokers[i].reg;
            }
        }
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"../../data/bin/BTCUSDT");
    bake(data);
    return 0;
}