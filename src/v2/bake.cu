#include "hip/hip_runtime.h"
#include "trade.h"

static __global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    brokers[workerNbr].cursor = cursor;
    tickBroker(&brokers[workerNbr]);
}

// #define TIME_START 700000
#define BAKE_MIN_BETS 500

long long current_timestamp() {
    struct timeval te;
    gettimeofday(&te, NULL);  // get current time
    long long milliseconds =
        te.tv_sec * 1000LL + te.tv_usec / 1000;  // calculate milliseconds
    return milliseconds;
}

static void bake(Data data) {
    int nbrThreads = 128;
    int nbrBlocks = 64;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    double maxBank = -999999999;
    double maxReg = 8;
    int totalMinutes = 0;
    for (int chien = 0; chien < 100000; chien++) {
        for (int i = 0; i < nbrWorkers; i++) {
            brokers[i] = newBroker(data);
        }
        long long timeStart = current_timestamp();
        for (int i = TIME_START; i < data.nbrMinutes; i++) {
            totalMinutes += 1;
            if (totalMinutes == 50000) {
                printf("perf: %lf\n",
                       (double)nbrWorkers /
                           (double)(current_timestamp() - timeStart));
            }
            // if (i % 100000 == 0) {
            //     printf("%d / 1300000  wokers : %d\n", i, nbrWorkers);
            // }
            applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
        }
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < nbrWorkers; i++) {
            // printf("BK: %-12.2lf\n\n", brokers[i].bank);
            if (brokers[i].bank > (maxBank * 0.95) && brokers[i].reg >= 10 &&
                brokers[i].nbrBets > BAKE_MIN_BETS) {
                // if (brokers[i].bank > 0 && brokers[i].reg >= maxReg &&
                // brokers[i].nbrBets > BAKE_MIN_BETS) {
                printSeed(&brokers[i].seed);
                printf("BK: %-8.02lf FEE: %-8.02lf NB: %-5d REG: %-5d\n\n",
                       brokers[i].bank, brokers[i].fees, brokers[i].nbrBets,
                       brokers[i].reg);
                maxBank = brokers[i].bank;
                maxReg = brokers[i].reg;
            }
        }
        // printf("DONE\n");
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"./BTCUSDT");
    bake(data);
    return 0;
}