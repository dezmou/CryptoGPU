#include "hip/hip_runtime.h"
#include "trade.h"

static __global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    Broker broker = brokers[workerNbr];
    broker.cursor = cursor;
    tickBroker(&broker);
    brokers[workerNbr] = broker;
}

#define TIME_START 700000
// #define TIME_START 0

static void bake(Data data) {
    int nbrThreads = 128;
    int nbrBlocks = 128;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    for (int i = 0; i < nbrWorkers; i++) {
        brokers[i] = newBroker(data);
    }
    for (int i = TIME_START; i < data.nbrMinutes; i++) {
        if (i % 1000 == 0) {
            printf("%d / 800000\n", i);
        }
        applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
    }
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    for (int i = 0; i < nbrWorkers; i++) {
        printf("CH: %-12.2lf BK: %-12.2lf\n", brokers[i].seed.chien,
               brokers[i].bank);
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"../../data/bin/BTCUSDT");
    bake(data);
    return 0;
}