
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    int nbrMinutes;
    Minute *minutes;
} Data;

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

void printMinute(Minute *minute) {
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf VOLUME: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close, minute->volume);
}

int main(){
    Data data = loadMinutes("./data");
    printMinute(&data.minutes[0]);
    return 0;
}