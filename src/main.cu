
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

typedef struct
{
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct
{
    int nbrMinutes;
    Minute *minutes;
} Data;

typedef struct
{
    long seed;
    long res;
} Worker;

__global__ void bake(Data data, Worker *workers)
{
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    workers[workerNbr].res = (workerNbr * 5 / 2 + 50 / 3 * 5) % 52 == 0 ? 1 : 0;
    // for (int i = 0; i < data.nbrMinutes * 0.1; i++)
    // {
    //     if (data.minutes[i].open > workers[workerNbr].seed)
    //     {

    //     }
    // }
}

Data loadMinutes(char *path)
{
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0)
    {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

void printMinute(Minute *minute)
{
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf VOLUME: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close, minute->volume);
}

void searchPike(Data data)
{
    printf("%d\n", data.nbrMinutes);
    int founds = 0;
    for (int i = 40; i < data.nbrMinutes - 40; i++)
    {
        double chien = data.minutes[i].open / data.minutes[i + 20].open;
        if (chien > 1.025 || chien < 0.975)
        {
            printf("%lf %d\n", chien, founds);
            founds += 1;
        }
    }
}

int main()
{
    Data data = loadMinutes("./data");
    // searchPike(data);

    int nbrX = 4096 * 8;
    int nbrY = 1024;
    int nbrThreads = nbrX * nbrY;

    // Worker *ramWorkers;
    // malloc(ramWorkers, nbrThreads * sizeof(Worker));

    Worker *workers;

    hipMalloc(&workers, nbrThreads * sizeof(Worker));
    // workers = (Worker *)malloc(nbrThreads * sizeof(Worker));


    // cudaMallocManaged(&workers, nbrThreads * sizeof(Worker));
    for (long i = 0; 1; i++)
    {
        bake<<<nbrX, nbrY>>>(data, workers);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        if (i % 10 == 0)
        {
            printf("DONE %ld - %ld B\n", i, i * nbrX * nbrY / 1000000000);
        }
        if (i * nbrX * nbrY / 1000000000 >= 100){
            // break;
        }
    }

    // for (long i = 0; 1; i++)
    // {
    //     long chien = i * 5 / 2 + 50 / 3 * 5;
    //     // workers[i % 2 == 0 ? 0 : 1] = chien;
    //     workers[1].res = chien % 52 == 0 ? 1 : 0;

    //     if (i % 1000000 == 0)
    //     {
    //         printf("DONE %ldM\n", i / 1000000);
    //     }
    // }

    // printMinute(&data.minutes[0]);
    return 0;
}