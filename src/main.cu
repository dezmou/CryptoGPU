
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

typedef struct {
    long time;
    double open;
    double high;
    double low;
    double close;
    double volume;
} Minute;

typedef struct {
    int nbrMinutes;
    Minute *minutes;
} Data;

typedef struct {
    long seed; 
    long res;
} Worker;

__global__ void bake(Data data, Worker *workers) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i =0; i < data.nbrMinutes; i++){
        if (data.minutes[i].open > workers[workerNbr].seed){
            workers[workerNbr].res = 1;
        }
    }
}

Data loadMinutes(char *path) {
    Data data;
    int fd = open(path, O_RDONLY);
    struct stat buf;
    fstat(fd, &buf);
    off_t size = buf.st_size;
    hipMallocManaged(&data.minutes, size);
    int rd = read(fd, data.minutes, size);
    if (rd <= 0) {
        printf("ERROR LOAD FILE\n");
        exit(0);
    }
    data.nbrMinutes = size / sizeof(Minute);
    return data;
}

void printMinute(Minute *minute) {
    printf("%ld OPEN: %-10.5lf HIGH: %-10.5lf LOW: %-10.5lf CLOSE: %-10.5lf VOLUME: %-10.5lf\n",
           minute->time, minute->open, minute->high, minute->low,
           minute->close, minute->volume);
}

int main(){
    Data data = loadMinutes("./data");

    int nbrX = 512;
    int nbrY = 512;
    int nbrThreads = nbrX * nbrY;

    Worker *workers;
    hipMallocManaged(&workers, nbrThreads * sizeof(Worker));
    for (int i=0; i < nbrThreads; i++){
        workers[i].seed = (double)i;
    }

    bake<<<512, 512>>>(data, workers);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    for (int i=0; i < nbrThreads; i++){
        printf("%ld\n", workers[i].res);
    }

    printMinute(&data.minutes[0]);
    return 0;
}