#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *y)
{
  // int index = threadIdx.x;
  // int stride = blockDim.x;
  printf("blockIdx.x: %d  threadIdx.x: %d gridDim.x: %d blockDim.x: %d\n", blockDim.x, threadIdx.x, gridDim.x, blockIdx.x);
}

Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void **) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

int main(void)
{
  int n = 1;
  float *y;
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&y, n*sizeof(float));
  y[0] = 5;
  add<<<10, 1>>>(n, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)

  // Free memory
  hipFree(y);
  
  return 0;
}