
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 200

#define NBR_COIN 162

#define NBR_COIN_CUDA 162
#define NBR_BLOCK 1024

#define NBR_HIGH_SCORE 20

// #define NBR_COIN_CUDA 4
// #define NBR_BLOCK 1

// #define NBR_COIN 1
// #define NBR_BLOCK 1

#define NBR_MINUTES 881003
#define AMOUNT_TEST 881003

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    int score;
    int minuteId;
    int coinId;
} Score;

typedef struct {
    Minute **minutes;
    Score highScores[NBR_HIGH_SCORE];
} Env;

Env env;

/**
 * Launch the great machine comparator
 * Comparing pourcent source with all other minutes
 */
__global__ void bake(Minute **source, int sourceCoinId, int cursor,
                     Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;
    double score = 0;
    for (int i = 0; i < SIT_SIZE; i++) {
        if (minutes[cursor + minuteId + i]->data[coinId].open == -1) {
            scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
            return;
        }
        double pourcent = minutes[cursor + minuteId + i]->data[coinId].open /
                          minutes[cursor + minuteId]->data[coinId].open * 100;
        score += fabs((source[i]->data[sourceCoinId].open) - (pourcent));
    }

    // printf("score : %12lf coinId: %4d minuteId : %3d test: %lf \n", score,
    //        coinId, minuteId + cursor,
    //        minutes[minuteId + cursor]->data[coinId].open);
    scores[NBR_COIN_CUDA * minuteId + coinId] = score;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Transform every value of a situation to a pourcentage from first value
 */
Minute **SituationToPourcent(int cursor) {
    Minute **result;
    hipMallocManaged(&result, sizeof(void *) * SIT_SIZE);
    for (int i = 0; i < SIT_SIZE; i++) {
        hipMallocManaged(&result[i], sizeof(Minute));
        result[i]->time = env.minutes[cursor + i]->time;
        for (int coinIndex = 0; coinIndex < NBR_COIN_CUDA; coinIndex++) {
            result[i]->data[coinIndex].close =
                env.minutes[cursor + i]->data[coinIndex].close /
                env.minutes[cursor]->data[coinIndex].close * 100;
            result[i]->data[coinIndex].high =
                env.minutes[cursor + i]->data[coinIndex].high /
                env.minutes[cursor]->data[coinIndex].high * 100;
            result[i]->data[coinIndex].low =
                env.minutes[cursor + i]->data[coinIndex].low /
                env.minutes[cursor]->data[coinIndex].low * 100;
            result[i]->data[coinIndex].open =
                env.minutes[cursor + i]->data[coinIndex].open /
                env.minutes[cursor]->data[coinIndex].open * 100;
            result[i]->data[coinIndex].volume =
                env.minutes[cursor + i]->data[coinIndex].volume /
                env.minutes[cursor + i]->data[coinIndex].volume * 100;
        }
    }
    return result;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor, int coinId) {
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE; i++) {
        dprintf(1, " %lf", env.minutes[i + cursor]->data[coinId].open);
    }
    dprintf(1, "\n");
}

/**
 * Compare Given situation with all history
 */
void bakeSituation(int cursor, int coinId) {
    int *scores;
    Minute **pourcent = SituationToPourcent(cursor);
    cursor += SIT_SIZE;  // avoiding compare source situation
    hipMallocManaged(&scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    for (int hi = 0; hi < NBR_HIGH_SCORE; hi++) {
        env.highScores[hi].score = 99999999;
        env.highScores[hi].minuteId = 0;
        env.highScores[hi].coinId = 0;
    }
    for (int bakeIndex = 0; cursor < 881003; bakeIndex++) {
        bake<<<NBR_BLOCK, NBR_COIN_CUDA>>>(pourcent, coinId, cursor,
                                           env.minutes, scores);
        hipDeviceSynchronize();
        for (int i = 0; i < NBR_BLOCK * NBR_COIN_CUDA; i++) {
            if (scores[i] != -1) {
                int minuteId = i / NBR_COIN;
                int coinId = i % NBR_COIN;

                // dprintf(2,
                //         "score : %12d coinId: %4d minuteid : %3d test:
                //         %lf\n", scores[i], coinId, minuteId + cursor,
                //         env.minutes[minuteId + cursor]->data[coinId].open);

                for (int highIndex = 0; highIndex < NBR_HIGH_SCORE;
                     highIndex++) {
                    if (scores[i] < env.highScores[highIndex].score) {
                        env.highScores[highIndex].score = scores[i];
                        env.highScores[highIndex].minuteId = minuteId + cursor;
                        env.highScores[highIndex].coinId = coinId;
                        dprintf(2,"score : %d\n", scores[i]);
                        break;
                    }
                }
                // if (scores[i] < 47) {
                //     dprintf(2, "score : %d coinId : %d\n time :", scores[i],
                //             coinId);
                //     printSituation(minuteId + cursor, coinId);
                //     // getchar();
                //     break;
                // }
            }
        }
        cursor += NBR_BLOCK;
        if (cursor % 100 == 0) {
            // printf("%d %d %d %d\n", env.highScores[0].score,
            //        env.highScores[1].score, env.highScores[2].score,
            //        env.highScores[3].score);
            dprintf(2, "cursor : %d\n", cursor);
            // getchar();
        }
        // getchar();
    }
    for (int highIndex = 0; highIndex < NBR_HIGH_SCORE; highIndex++) {
        printSituation(env.highScores[highIndex].minuteId, env.highScores[highIndex].coinId);
        getchar();
    }
}

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

/**
 * do something with the score of a minute
 */
void onScore() {}

int main() {
    clear();
    env.minutes = loadHistory(0, AMOUNT_TEST);
    dprintf(2, "ready\n");
    getchar();
    int cursor = 405000;
    printSituation(cursor, 25);
    getchar();
    bakeSituation(cursor, 25);

    return 0;
}