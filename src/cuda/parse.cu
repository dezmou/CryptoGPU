
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 1000
#define NBR_COIN 162
#define NBR_MINUTES 881003
#define AMOUNT_TEST 500000
#define NBR_BLOCK 128

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    Minute **minutes;
    int endIndex;

} Situation;

typedef struct {
    Minute **minutes;
} Env;

Env env;

__global__ void test(Minute **minutes) {
    // int coinId = threadIdx.x;
    // int minuteId = blockDim.x;

    // printf("")

    // for (int i = 0; i < AMOUNT_TEST; i++) {
    //     printf("%lf - %lf\n", minutes[i]->data[3].open,
    //            minutes[i]->data[3].volume);
    // }
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void **) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Compare Given situation with all history
 */
void bakeSituation() {}

/**
 * Export situation to external program
 */
void printSituation(Situation *sit) {
    int index = sit->endIndex - SIT_SIZE;
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE; i++) {
        dprintf(1, " %lf", sit->minutes[index + i * 10]->data[3].open);
    }
    dprintf(1,"\n");
}

void clear(){
    dprintf(1, "#CLS\n");
}

int main() {
    clear();
    env.minutes = loadHistory(0, AMOUNT_TEST);
    dprintf(2,"ready");
    Situation sit;
    sit.minutes = env.minutes;
    for (int i=0 ; i < 400; i++){
        sit.endIndex = 2000 + (i * 400);
        printSituation(&sit);
        getchar();
        clear();
    }
    // int nbrIteration = NBR_MINUTES / NBR_BLOCK;
    // test<<<NBR_BLOCK, NBR_COIN>>>(env.minutes);
    // cudaDeviceSynchronize();
    return 0;
}