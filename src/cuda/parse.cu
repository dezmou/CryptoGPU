
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 130

#define NBR_COIN 162

#define NBR_COIN_CUDA 162
#define NBR_BLOCK 1024

#define NBR_HIGH_SCORE 50

// #define NBR_COIN_CUDA 4
// #define NBR_BLOCK 1

// #define NBR_COIN 1
// #define NBR_BLOCK 1

#define NBR_MINUTES 881003
#define AMOUNT_TEST 881003

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    int score;
    int minuteId;
    int coinId;
} Score;

typedef struct {
    Score highScores[NBR_HIGH_SCORE];
    double *guessed;

    /**Cuda memory */
    Minute **minutes;  // all history
    Minute **srcPourcent;
    int *scores;
} Env;

Env env;

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

/**
 * Launch the great machine comparator
 * Comparing pourcent source with all other minutes
 */
__global__ void bake(Minute **source, int sourceCoinId, int cursor,
                     Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;
    double score = 0;
    if (minutes[cursor + minuteId]->data[coinId].open < 0.000220) {
        scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
        return;
    }
    for (int i = 0; i < SIT_SIZE; i++) {
        if (minutes[cursor + minuteId + i]->data[coinId].open == -1) {
            scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
            return;
        }
        double pourcent = minutes[cursor + minuteId + i]->data[coinId].open /
                          minutes[cursor + minuteId]->data[coinId].open * 100;
        score +=
            fabs(fabs(source[i]->data[sourceCoinId].open) - fabs(pourcent));
    }

    // printf("score : %12lf coinId: %4d minuteId : %3d test: %lf \n", score,
    //        coinId, minuteId + cursor,
    //        minutes[minuteId + cursor]->data[coinId].open);
    scores[NBR_COIN_CUDA * minuteId + coinId] = score;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Transform every value of a situation to a pourcentage from first value
 */
Minute **SituationToPourcent(int cursor) {
    for (int i = 0; i < SIT_SIZE; i++) {
        env.srcPourcent[i]->time = env.minutes[cursor + i]->time;
        for (int coinIndex = 0; coinIndex < NBR_COIN_CUDA; coinIndex++) {
            env.srcPourcent[i]->data[coinIndex].close =
                env.minutes[cursor + i]->data[coinIndex].close /
                env.minutes[cursor]->data[coinIndex].close * 100;
            env.srcPourcent[i]->data[coinIndex].high =
                env.minutes[cursor + i]->data[coinIndex].high /
                env.minutes[cursor]->data[coinIndex].high * 100;
            env.srcPourcent[i]->data[coinIndex].low =
                env.minutes[cursor + i]->data[coinIndex].low /
                env.minutes[cursor]->data[coinIndex].low * 100;
            env.srcPourcent[i]->data[coinIndex].open =
                env.minutes[cursor + i]->data[coinIndex].open /
                env.minutes[cursor]->data[coinIndex].open * 100;
            env.srcPourcent[i]->data[coinIndex].volume =
                env.minutes[cursor + i]->data[coinIndex].volume /
                env.minutes[cursor + i]->data[coinIndex].volume * 100;
        }
    }
    return env.srcPourcent;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor, int coinId) {
    dprintf(2, "sit : %lf coinId : %d\n", env.minutes[cursor]->time, coinId);
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE * 2; i++) {
        dprintf(2, " %lf", env.minutes[i + cursor]->data[coinId].open);
        dprintf(1, " %lf", env.minutes[i + cursor]->data[coinId].open);
    }
    dprintf(1, "\n");
}

/**
 * Compare Given situation with all history
 */
void bakeSituation(int cursor, int coinId) {
    // score
    int *scores = env.scores;
    dprintf(2, "11\n");
    Minute **pourcent = SituationToPourcent(cursor);
    // cursor += SIT_SIZE;  // avoiding compare source situation
    cursor = 0;
    dprintf(2, "1\n");
    dprintf(2, "2\n");
    for (int hi = 0; hi < NBR_HIGH_SCORE; hi++) {
        env.highScores[hi].score = 99999999;
        env.highScores[hi].minuteId = 0;
        env.highScores[hi].coinId = 0;
    }
    for (int bakeIndex = 0; cursor < 870000; bakeIndex++) {
        bake<<<NBR_BLOCK, NBR_COIN_CUDA>>>(pourcent, coinId, cursor,
                                           env.minutes, scores);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < NBR_BLOCK * NBR_COIN_CUDA; i++) {
            if (scores[i] != -1) {
                int minuteId = i / NBR_COIN;
                int coinId = i % NBR_COIN;

                // dprintf(2,
                //         "score : %12d coinId: %4d minuteid : %3d test:
                //         %lf\n", scores[i], coinId, minuteId + cursor,
                //         env.minutes[minuteId + cursor]->data[coinId].open);

                for (int highIndex = 0; highIndex < NBR_HIGH_SCORE;
                     highIndex++) {
                    if (scores[i] < env.highScores[highIndex].score) {
                        env.highScores[highIndex].score = scores[i];
                        env.highScores[highIndex].minuteId = minuteId + cursor;
                        env.highScores[highIndex].coinId = coinId;
                        i += NBR_COIN_CUDA * 50;
                        break;
                    }
                }
                // if (found) {
                //     break;
                // }
                // if (scores[i] < 47) {
                //     dprintf(2, "score : %d coinId : %d\n time :", scores[i],
                //             coinId);
                //     printSituation(minuteId + cursor, coinId);
                //     // getchar();
                //     break;
                // }
            }
        }
        cursor += NBR_BLOCK;
        if (cursor % 100 == 0) {
            // dprintf(2, "cursor : %d\n", cursor);
            // getchar();
        }
        // getchar();
    }
    dprintf(2, "Done\n");
    // getchar();

    // clear();
    for (int highIndex = 0; highIndex < NBR_HIGH_SCORE-1; highIndex++) {
        getchar();
        printSituation(env.highScores[highIndex].minuteId,
                       env.highScores[highIndex].coinId);
    }
}

// void makeGuess() {
//     for (int i = 0; i < SIT_SIZE; i++) {
//         for (int highIndex = 0; highIndex < NBR_HIGH_SCORE; highIndex++) {
//             env.highScores[highIndex].minuteId + SIT_SIZE;
//             env.highScores[highIndex].coinId;
//         }
//     }
// }

// /**
//  * do something with the score of a minute
//  */
// void onScore() {}

void initMem() {
    hipMallocManaged(&env.srcPourcent, sizeof(void *) * SIT_SIZE);
    for (int i = 0; i < SIT_SIZE; i++) {
        hipMallocManaged(&env.srcPourcent[i], sizeof(Minute));
    }
    hipMallocManaged(&env.scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    env.guessed = (double *)malloc(sizeof(double) * SIT_SIZE);
}

int main() {
    env.minutes = loadHistory(0, AMOUNT_TEST);
    initMem();
    int cur = 0;
    while (1) {
        // dprintf(2, "ready\n");
        int cursor = 457100 + cur;
        clear();
        printSituation(cursor, 98);
        dprintf(2, "READY\n");
        bakeSituation(cursor, 98);
        exit(0);
        // makeGuess();
        cur += SIT_SIZE / 2;
    }
    return 0;
}