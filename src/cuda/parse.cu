
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 400

#define NBR_COIN 162
// #define NBR_BLOCK 128

#define NBR_COIN_CUDA 162
#define NBR_BLOCK 128

// #define NBR_COIN 1
// #define NBR_BLOCK 1

#define NBR_MINUTES 881003
#define AMOUNT_TEST 881003

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    Minute **minutes;
} Env;

Env env;

/**
 * Launch the great machine comparator
 * Comparing pourcent source with all other minutes
 */
__global__ void bake(Minute **source, int sourceCoinId, int cursor,
                     Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;
    double score = 0;
    for (int i = 0; i < SIT_SIZE; i++) {
        if (minutes[cursor + minuteId + i]->data[coinId].open == -1) {
            scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
            return;
        }
        double pourcent = minutes[cursor + minuteId + i]->data[coinId].open /
                          minutes[cursor + minuteId]->data[coinId].open * 100;
        score += fabs((source[i]->data[sourceCoinId].open) - (pourcent));
    }

    // printf("score : %12lf coinId: %4d minuteId : %3d test: %lf \n", score,
    //        coinId, minuteId + cursor,
    //        minutes[minuteId + cursor]->data[coinId].open);

    scores[NBR_COIN_CUDA * minuteId + coinId] = score;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Transform every value of a situation to a pourcentage from first value
 */
Minute **SituationToPourcent(int cursor) {
    Minute **result;
    hipMallocManaged(&result, sizeof(void *) * SIT_SIZE);
    for (int i = 0; i < SIT_SIZE; i++) {
        hipMallocManaged(&result[i], sizeof(Minute));
        result[i]->time = env.minutes[cursor + i]->time;
        for (int coinIndex = 0; coinIndex < NBR_COIN_CUDA; coinIndex++) {
            result[i]->data[coinIndex].close =
                env.minutes[cursor + i]->data[coinIndex].close /
                env.minutes[cursor]->data[coinIndex].close * 100;
            result[i]->data[coinIndex].high =
                env.minutes[cursor + i]->data[coinIndex].high /
                env.minutes[cursor]->data[coinIndex].high * 100;
            result[i]->data[coinIndex].low =
                env.minutes[cursor + i]->data[coinIndex].low /
                env.minutes[cursor]->data[coinIndex].low * 100;
            result[i]->data[coinIndex].open =
                env.minutes[cursor + i]->data[coinIndex].open /
                env.minutes[cursor]->data[coinIndex].open * 100;
            result[i]->data[coinIndex].volume =
                env.minutes[cursor + i]->data[coinIndex].volume /
                env.minutes[cursor + i]->data[coinIndex].volume * 100;
        }
    }
    return result;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor, int coinId) {
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE; i++) {
        dprintf(1, " %lf", env.minutes[i + cursor]->data[coinId].open);
    }
    dprintf(1, "\n");
}

/**
 * Compare Given situation with all history
 */
int *bakeSituation(int cursor, int coinId) {
    int *scores;
    Minute **pourcent = SituationToPourcent(cursor);
    cursor += SIT_SIZE;  // avoiding compare source situation
    hipMallocManaged(&scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    for (int bakeIndex = 0; cursor < 881003; bakeIndex++) {
        bake<<<NBR_BLOCK, NBR_COIN_CUDA>>>(pourcent, coinId, cursor,
                                           env.minutes, scores);
        hipDeviceSynchronize();
        for (int i = 0; i < NBR_BLOCK * NBR_COIN_CUDA; i++) {
            if (scores[i] != -1) {
                int minuteId = i / NBR_COIN;
                int coinId = i % NBR_COIN;

                // dprintf(2,
                //         "score : %12d coinId: %4d minuteid : %3d test:
                //         %lf\n", scores[i], coinId, minuteId + cursor,
                //         env.minutes[minuteId + cursor]->data[coinId].open);

                if (scores[i] < 1300) {
                    printSituation(minuteId + cursor, coinId);
                    getchar();
                }
            }
        }
        cursor += NBR_BLOCK;
        if (cursor % 100 == 0) {
            dprintf(2, "cursor : %d\n", cursor);
        }
        // getchar();
    }
    return scores;
}

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

/**
 * do something with the score of a minute
 */
void onScore() {}

int main() {
    clear();
    env.minutes = loadHistory(0, AMOUNT_TEST);
    dprintf(2, "ready\n");
    int cursor = 0;
    printSituation(cursor, 3);
    getchar();
    int *scores = bakeSituation(cursor, 3);

    return 0;
}