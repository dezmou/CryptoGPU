#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define CBLACK "\33[30m"
#define CRED "\33[31m"
#define CGREEN "\33[32m"
#define CWHITE "\33[37m"

#define SIT_SIZE 130
#define NBR_COIN 162
#define NBR_COIN_CUDA 162
#define NBR_BLOCK 1024
#define NBR_HIGH_SCORE 10
#define MIN_PRICE 0.000220
#define TIME_GUESS 10
#define COIN_TEST 98
#define AMOUNT_BET 100
#define MIN_POURCENT_GUESS 0.001
#define NBR_MINUTES 881003
#define AMOUNT_TEST 881003

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    int score;
    int minuteId;
    int coinId;
} Score;

typedef struct {
    Score highScores[NBR_HIGH_SCORE];
    double *guessed;

    /**Cuda memory */
    Minute **minutes;  // all history
    Minute **srcPourcent;
    int *scores;
} Env;

typedef struct {
    int cursor;
    int coinId;
} Situation;

Env env;

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

/**
 * Launch the great machine comparator
 * Comparing pourcent source with all other minutes
 */
__global__ void bake(Minute **source, int sourceCoinId, int cursor,
                     Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;
    double score = 0;
    if (minutes[cursor + minuteId]->data[coinId].open < MIN_PRICE) {
        scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
        return;
    }
    for (int i = 0; i < SIT_SIZE; i++) {
        if (minutes[cursor + minuteId + i]->data[coinId].open == -1) {
            scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
            return;
        }
        double pourcent = minutes[cursor + minuteId + i]->data[coinId].open /
                          minutes[cursor + minuteId]->data[coinId].open * 100;
        score +=
            fabs(fabs(source[i]->data[sourceCoinId].open) - fabs(pourcent));
    }

    // printf("score : %12lf coinId: %4d minuteId : %3d test: %lf \n", score,
    //        coinId, minuteId + cursor,
    //        minutes[minuteId + cursor]->data[coinId].open);
    scores[NBR_COIN_CUDA * minuteId + coinId] = score;
}

/**
 * Generate a random number
 */
int random_number(int min_num, int max_num) {
    int result = (rand() % (max_num - min_num)) + min_num;
    return result;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Transform every value of a situation to a pourcentage from first value
 */
Minute **SituationToPourcent(int cursor) {
    for (int i = 0; i < SIT_SIZE; i++) {
        env.srcPourcent[i]->time = env.minutes[cursor + i]->time;
        for (int coinIndex = 0; coinIndex < NBR_COIN_CUDA; coinIndex++) {
            env.srcPourcent[i]->data[coinIndex].close =
                env.minutes[cursor + i]->data[coinIndex].close /
                env.minutes[cursor]->data[coinIndex].close * 100;
            env.srcPourcent[i]->data[coinIndex].high =
                env.minutes[cursor + i]->data[coinIndex].high /
                env.minutes[cursor]->data[coinIndex].high * 100;
            env.srcPourcent[i]->data[coinIndex].low =
                env.minutes[cursor + i]->data[coinIndex].low /
                env.minutes[cursor]->data[coinIndex].low * 100;
            env.srcPourcent[i]->data[coinIndex].open =
                env.minutes[cursor + i]->data[coinIndex].open /
                env.minutes[cursor]->data[coinIndex].open * 100;
            env.srcPourcent[i]->data[coinIndex].volume =
                env.minutes[cursor + i]->data[coinIndex].volume /
                env.minutes[cursor + i]->data[coinIndex].volume * 100;
        }
    }
    return env.srcPourcent;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor, int coinId) {
    dprintf(2, "sit : %lf coinId : %d\n", env.minutes[cursor]->time, coinId);
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE * 2; i++) {
        dprintf(2, " %lf", env.minutes[i + cursor]->data[coinId].open);
        dprintf(1, " %lf", env.minutes[i + cursor]->data[coinId].open);
    }
    dprintf(1, "\n");
}

/**
 * Compare Given situation with all history
 */
void bakeSituation(int cursor, int baseCoinId) {
    // score
    int *scores = env.scores;
    int baseCursor = cursor;
    Minute **pourcent = SituationToPourcent(cursor);
    // cursor += SIT_SIZE;  // avoiding compare source situation
    cursor = 0;
    for (int hi = 0; hi < NBR_HIGH_SCORE; hi++) {
        env.highScores[hi].score = 99999999;
        env.highScores[hi].minuteId = 0;
        env.highScores[hi].coinId = 0;
    }
    for (int bakeIndex = 0; cursor < 870000; bakeIndex++) {
        bake<<<NBR_BLOCK, NBR_COIN_CUDA>>>(pourcent, baseCoinId, cursor,
                                           env.minutes, scores);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < NBR_BLOCK * NBR_COIN_CUDA; i++) {
            if (scores[i] != -1) {
                int minuteId = i / NBR_COIN;
                int coinId = i % NBR_COIN;
                if (abs((minuteId + cursor) - baseCursor) < (SIT_SIZE * 5)) {
                    continue;
                }

                // dprintf(2,
                //         "score : %12d coinId: %4d minuteid : %3d test:
                //         %lf\n", scores[i], coinId, minuteId + cursor,
                //         env.minutes[minuteId + cursor]->data[coinId].open);

                for (int highIndex = 0; highIndex < NBR_HIGH_SCORE;
                     highIndex++) {
                    if (scores[i] < env.highScores[highIndex].score) {
                        env.highScores[highIndex].score = scores[i];
                        env.highScores[highIndex].minuteId = minuteId + cursor;
                        env.highScores[highIndex].coinId = coinId;
                        i += NBR_COIN_CUDA * 50;
                        break;
                    }
                }
                // if (found) {
                //     break;
                // }
                // if (scores[i] < 47) {
                //     dprintf(2, "score : %d coinId : %d\n time :", scores[i],
                //             coinId);
                //     printSituation(minuteId + cursor, coinId);
                //     // getchar();
                //     break;
                // }
            }
        }
        cursor += NBR_BLOCK;
        // if (cursor % 100 == 0) {
        //     // dprintf(2, "cursor : %d\n", cursor);
        //     // getchar();
        // }
        // getchar();
    }
    // dprintf(2, "Done\n");
    // getchar();

    // clear();
    // for (int highIndex = 0; highIndex < NBR_HIGH_SCORE - 1; highIndex++) {
    //     getchar();
    //     printSituation(env.highScores[highIndex].minuteId,
    //                    env.highScores[highIndex].coinId);
    // }
}

/**
 * Return the guessed percentage of change from situation to TIME_GUESS
 */
double makeNextGuess() {
    double pred = 0;
    for (int highIndex = 0; highIndex < NBR_HIGH_SCORE; highIndex++) {
        // env.highScores[highIndex].minuteId + SIT_SIZE;
        // env.highScores[highIndex].coinId;
        double start =
            env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE]
                ->data[env.highScores[highIndex].coinId]
                .open;
        double end = env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE +
                                 TIME_GUESS]
                         ->data[env.highScores[highIndex].coinId]
                         .open;
        pred += 100 - (start / end * 100);
    }
    pred = pred / NBR_HIGH_SCORE;
    return pred;
}

/**
 * Get real next pourcent of given situation
 */
double getRealNext(int minuteId, int coinId) {
    double start = env.minutes[minuteId + SIT_SIZE]->data[coinId].open;
    double end =
        env.minutes[minuteId + SIT_SIZE + TIME_GUESS]->data[coinId].open;
    return 100 - (start / end * 100);
}

// /**
//  * do something with the score of a minute
//  */
// void onScore() {}

void initMem() {
    hipMallocManaged(&env.srcPourcent, sizeof(void *) * SIT_SIZE);
    for (int i = 0; i < SIT_SIZE; i++) {
        hipMallocManaged(&env.srcPourcent[i], sizeof(Minute));
    }
    hipMallocManaged(&env.scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    env.guessed = (double *)malloc(sizeof(double) * SIT_SIZE);
}

Situation getRandomSituation() {
    Situation res;
    int last = 0;
    while (1) {
        res.cursor = random_number(200000, NBR_MINUTES - 1000);
        if (res.cursor == last) {
            printf("AH NON\n");
        }
        last = res.cursor;
        res.coinId = random_number(0, NBR_COIN_CUDA);
        if (env.minutes[res.cursor]->data[res.coinId].open != -1 &&
            env.minutes[res.cursor]->data[res.coinId].open > MIN_PRICE) {
            return res;
        }
        usleep(1000);
    }
}

int main() {
    srand(time(NULL));
    env.minutes = loadHistory(0, AMOUNT_TEST);
    initMem();
    int cur = 0;
    double bank = 1000;
    while (1) {
        // dprintf(2, "ready\n");
        // int cursor = 397100 + cur;
        // int cursor = random_number(397100, 500000);
        // clear();
        // printSituation(cursor, COIN_TEST);
        // dprintf(2, "READY\n");
        Situation sit = getRandomSituation();
        bakeSituation(sit.cursor, sit.coinId);
        double pred = makeNextGuess();
        double real = getRealNext(sit.cursor, sit.coinId);
        if (abs(real) > 5) {
            continue;
        }
        printf(
            "Time : %d | Cursor : %8d | CoinId : %4d | Pred : %10lf | Real : "
            "%10lf | BANK : %12lf |",
            (int)env.minutes[sit.cursor + SIT_SIZE]->time, sit.cursor,
            sit.coinId, pred, real, bank);
        if (abs(pred) > MIN_POURCENT_GUESS) {
            if (pred * real > 0) {
                bank += abs(real) * AMOUNT_BET;
                // printf("%sWON  %s ", CGREEN, CWHITE);
                bank += -(AMOUNT_BET * 0.002);
            } else {
                // printf("%sLOST %s ", CRED, CWHITE);
                bank -= abs(real) * AMOUNT_BET;
                bank += -(AMOUNT_BET * 0.002);
            }
        }
        printf("\n");
        fflush(stdout);
        // write(1,"\n", 1);
        // exit(0);
        // cur += SIT_SIZE / 2;
    }
    return 0;
}