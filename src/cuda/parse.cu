
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 1000

#define NBR_COIN 162
#define NBR_BLOCK 128

// #define NBR_COIN 1
// #define NBR_BLOCK 1

#define NBR_MINUTES 881003
#define AMOUNT_TEST 1000

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    Minute **minutes;
} Env;

Env env;

__global__ void bake(int cursor, Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;

    // for (int i=0; i < SIT_SIZE; i++){
    //     printf("%lf\n", minutes[minuteId + i]->data[3].open);
    // }

    // for (int i = 0; i < AMOUNT_TEST; i++) {
    //     printf("%lf - %lf\n", minutes[i]->data[3].open,
    //            minutes[i]->data[3].volume);
    // }

    scores[ NBR_COIN * minuteId + coinId] = 69;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Compare Given situation with all history
 */
int *bakeSituation(int cursor) {
    int *scores;
    hipMallocManaged(&scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    // int nbrIteration = NBR_MINUTES / NBR_BLOCK;
    bake<<<NBR_BLOCK, NBR_COIN>>>(cursor, env.minutes, scores);
    hipDeviceSynchronize();
    return scores;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor) {
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE; i++) {
        dprintf(1, " %lf", env.minutes[cursor + i]->data[3].open);
    }
    dprintf(1, "\n");
}

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

int main() {
    // clear();
    env.minutes = loadHistory(0, AMOUNT_TEST);
    dprintf(2, "ready\n");
    int cursor = 0;
    int *scores = bakeSituation(cursor);

    for (int i = 0; i < NBR_BLOCK * NBR_COIN; i++) {
        dprintf(2, "%d ", scores[i]);
    }
    dprintf(2,"\n");


    // for (int i = 0; i < 400; i++) {
    //     clear();
    //     printSituation(cursor);
    //     cursor += 5000;
    //     getchar();
    // }

    return 0;
}