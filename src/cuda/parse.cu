
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 200
#define NBR_COIN 162
#define NBR_MINUTES 881003
#define AMOUNT_TEST 100000
#define NBR_BLOCK 128

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    Minute **minutes;
    int endIndex;

} Situation;

typedef struct {
    Minute **minutes;
} Env;

Env env;

__global__ void test(Minute **minutes) {
    // int coinId = threadIdx.x;
    // int minuteId = blockDim.x;

    // printf("")

    // for (int i = 0; i < AMOUNT_TEST; i++) {
    //     printf("%lf - %lf\n", minutes[i]->data[3].open,
    //            minutes[i]->data[3].volume);
    // }
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void **) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Compare Given situation with all history
 */
void bakeSituation() {}

/**
 * Export situation to external program
 */
void printSituation() {}

int main() {
    // printf("CHIEN\n");
    // env.minutes = loadHistory(0, AMOUNT_TEST);
    // int nbrIteration = NBR_MINUTES / NBR_BLOCK;
    // test<<<NBR_BLOCK, NBR_COIN>>>(env.minutes);
    // cudaDeviceSynchronize();
    // printf("done\n");
    dprintf(1, "#LE CHIEN\n");
    dprintf(1, "#LE RIEN\n");
    dprintf(1, "#LE CASSOULET\n");
    getchar();
    dprintf(1, "#LE MOULINSARD\n");
    return 0;
}