#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define CBLACK "\33[30m"
#define CRED "\33[31m"
#define CGREEN "\33[32m"
#define CWHITE "\33[37m"

#define SIT_SIZE 500
#define NBR_COIN 162
#define NBR_COIN_CUDA 162
#define NBR_BLOCK 1024
#define NBR_HIGH_SCORE 50
#define MIN_PRICE 0.000620
#define TIME_GUESS 100
#define COIN_TEST 98
#define AMOUNT_BET 100
#define MIN_POURCENT_GUESS 0.001
#define NBR_MINUTES 881003
#define AMOUNT_TEST 881003

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

typedef struct {
    int score;
    int minuteId;
    int coinId;
} Score;

typedef struct {
    Score highScores[NBR_HIGH_SCORE];
    double *guessed;

    /**Cuda memory */
    Minute **minutes;  // all history
    Minute **srcPourcent;
    int *scores;
} Env;

typedef struct {
    int cursor;
    int coinId;
} Situation;

Env env;

/**
 * Clear visual field
 */
void clear() { dprintf(1, "#CLS\n"); }

/**
 * Launch the great machine comparator
 * Comparing pourcent source with all other minutes
 */
__global__ void bake(Minute **source, int sourceCoinId, int cursor,
                     Minute **minutes, int *scores) {
    int coinId = threadIdx.x;
    int minuteId = blockIdx.x;
    double score = 0;
    if (minutes[cursor + minuteId]->data[coinId].open < MIN_PRICE) {
        scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
        return;
    }
    for (int i = 0; i < SIT_SIZE; i++) {
        if (minutes[cursor + minuteId + i]->data[coinId].open == -1) {
            scores[NBR_COIN_CUDA * minuteId + coinId] = -1;
            return;
        }
        double pourcent = minutes[cursor + minuteId + i]->data[coinId].open /
                          minutes[cursor + minuteId]->data[coinId].open * 100;
        score +=
            fabs(fabs(source[i]->data[sourceCoinId].open) - fabs(pourcent));
    }
    scores[NBR_COIN_CUDA * minuteId + coinId] = score;
}

/**
 * Generate a random number
 */
int random_number(int min_num, int max_num) {
    int result = (rand() % (max_num - min_num)) + min_num;
    return result;
}

/**
 * Load history in RAM and VRAM
 */
Minute **loadHistory(int start, int amount) {
    int fd = open("../data/bin/full", O_RDONLY);
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void *) * amount);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], sizeof(Minute)) < 1 || i == AMOUNT_TEST) break;
    }
    return minutes;
}

/**
 * Transform every value of a situation to a pourcentage from first value
 */
Minute **SituationToPourcent(int cursor) {
    for (int i = 0; i < SIT_SIZE; i++) {
        env.srcPourcent[i]->time = env.minutes[cursor + i]->time;
        for (int coinIndex = 0; coinIndex < NBR_COIN_CUDA; coinIndex++) {
            env.srcPourcent[i]->data[coinIndex].close =
                env.minutes[cursor + i]->data[coinIndex].close /
                env.minutes[cursor]->data[coinIndex].close * 100;
            env.srcPourcent[i]->data[coinIndex].high =
                env.minutes[cursor + i]->data[coinIndex].high /
                env.minutes[cursor]->data[coinIndex].high * 100;
            env.srcPourcent[i]->data[coinIndex].low =
                env.minutes[cursor + i]->data[coinIndex].low /
                env.minutes[cursor]->data[coinIndex].low * 100;
            env.srcPourcent[i]->data[coinIndex].open =
                env.minutes[cursor + i]->data[coinIndex].open /
                env.minutes[cursor]->data[coinIndex].open * 100;
            env.srcPourcent[i]->data[coinIndex].volume =
                env.minutes[cursor + i]->data[coinIndex].volume /
                env.minutes[cursor + i]->data[coinIndex].volume * 100;
        }
    }
    return env.srcPourcent;
}

/**
 * Export situation to external program
 */
void printSituation(int cursor, int coinId) {
    dprintf(2, "sit : %lf coinId : %d\n", env.minutes[cursor]->time, coinId);
    dprintf(1, "#SIT");
    for (int i = 0; i < SIT_SIZE; i++) {
        dprintf(2, " %lf", env.minutes[i + cursor]->data[coinId].open);
        dprintf(1, " %lf", env.minutes[i + cursor]->data[coinId].open);
    }
    dprintf(1, "\n");
}

/**
 * Compare Given situation with all history
 */
void bakeSituation(int cursor, int baseCoinId) {
    // score
    int *scores = env.scores;
    int baseCursor = cursor;
    Minute **pourcent = SituationToPourcent(cursor);
    cursor = 0;
    for (int hi = 0; hi < NBR_HIGH_SCORE; hi++) {
        env.highScores[hi].score = 99999999;
        env.highScores[hi].minuteId = 0;
        env.highScores[hi].coinId = 0;
    }
    for (int bakeIndex = 0; cursor < 870000; bakeIndex++) {
        bake<<<NBR_BLOCK, NBR_COIN_CUDA>>>(pourcent, baseCoinId, cursor,
                                           env.minutes, scores);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < NBR_BLOCK * NBR_COIN_CUDA; i++) {
            if (scores[i] != -1) {
                int minuteId = i / NBR_COIN;
                int coinId = i % NBR_COIN;
                if (abs((minuteId + cursor) - baseCursor) < (SIT_SIZE * 5)) {
                    continue;
                }
                for (int highIndex = 0; highIndex < NBR_HIGH_SCORE;
                     highIndex++) {
                    if (scores[i] < env.highScores[highIndex].score) {
                        env.highScores[highIndex].score = scores[i];
                        env.highScores[highIndex].minuteId = minuteId + cursor;
                        env.highScores[highIndex].coinId = coinId;
                        i += NBR_COIN_CUDA * 50;
                        break;
                    }
                }
            }
        }
        cursor += NBR_BLOCK;
    }
}

/**
 * Return the guessed percentage of change from situation to TIME_GUESS
 */
double makeNextGuess() {
    double pred = 0;
    for (int highIndex = 0; highIndex < NBR_HIGH_SCORE; highIndex++) {
        double start =
            env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE]
                ->data[env.highScores[highIndex].coinId]
                .open;
        double end = env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE +
                                 TIME_GUESS]
                         ->data[env.highScores[highIndex].coinId]
                         .open;
        pred += 100 - (start / end * 100);
    }
    pred = pred / NBR_HIGH_SCORE;
    return pred;
}

/**
 * Get real next pourcent of given situation
 */
double getRealNext(int minuteId, int coinId) {
    double start = env.minutes[minuteId + SIT_SIZE]->data[coinId].open;
    double end =
        env.minutes[minuteId + SIT_SIZE + TIME_GUESS]->data[coinId].open;
    return 100 - (start / end * 100);
}

void initMem() {
    hipMallocManaged(&env.srcPourcent, sizeof(void *) * SIT_SIZE);
    for (int i = 0; i < SIT_SIZE; i++) {
        hipMallocManaged(&env.srcPourcent[i], sizeof(Minute));
    }
    hipMallocManaged(&env.scores, sizeof(int) * NBR_BLOCK * NBR_COIN);
    env.guessed = (double *)malloc(sizeof(double) * SIT_SIZE);
}

Situation getRandomSituation() {
    Situation res;
    int last = 0;
    while (1) {
        res.cursor = random_number(200000, NBR_MINUTES - 1000);
        last = res.cursor;
        res.coinId = random_number(0, NBR_COIN_CUDA);
        if (env.minutes[res.cursor]->data[res.coinId].open != -1 &&
            env.minutes[res.cursor]->data[res.coinId].open > MIN_PRICE) {
            return res;
        }
        usleep(1000);
    }
}

void printInfos(Situation sit) {
    FILE *fp;
    fp = fopen("tmp", "w");
    fprintf(fp,"%d;%d(", sit.coinId, sit.cursor);
    for (int i = 20; i < 220; i += 20) {
        double start =
            env.minutes[sit.cursor + SIT_SIZE]->data[sit.coinId].open;
        double end =
            env.minutes[sit.cursor + SIT_SIZE + i]->data[sit.coinId].open;
        double pred = 100 - (start / end * 100);
        fprintf(fp,"%lf;", pred);
    }
    fprintf(fp,")-->");
    for (int highIndex = 0; highIndex < NBR_HIGH_SCORE; highIndex++) {
        fprintf(fp,"%d;%d(", env.highScores[highIndex].coinId,
               env.highScores[highIndex].minuteId);
        for (int i = 20; i < 220; i += 20) {
            double start =
                env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE]
                    ->data[env.highScores[highIndex].coinId]
                    .open;
            double end =
                env.minutes[env.highScores[highIndex].minuteId + SIT_SIZE + i]
                    ->data[env.highScores[highIndex].coinId]
                    .open;
            double pred = 100 - (start / end * 100);
            fprintf(fp,"%lf;", pred);
        }
        fprintf(fp,")|");
    }
    fprintf(fp,"\n");
    fclose(fp);
}

int main() {
    srand(time(NULL));
    env.minutes = loadHistory(0, AMOUNT_TEST);
    initMem();
    Situation sit;
    Data *tmp = (Data*)malloc(sizeof(Data) * 500);
    double last = -1;
    while (1) {
        int fd = open("./actual",O_RDONLY);
        if (fd < 1){
            printf("CUDA SLEEP 1\n");
            sleep(1);
            continue;
        }
        int res = read(fd, tmp, sizeof(Data) * 500);
        if (last == tmp[0].open){
            printf("CUDA SLEEP 2\n");
            sleep(1);
            continue;
        }
        printf("PROCESSING !\n");
        for (int i=0; i < 500; i++){
            env.minutes[i]->data[0].open = tmp[i].open;
        }
        bakeSituation(0, 0);
        printInfos(sit);
        last = tmp[0].open;
        sleep(1);
        
        // break;
        // Situation sit = getRandomSituation();
        // bakeSituation(sit.cursor, sit.coinId);
    }
    return 0;
}