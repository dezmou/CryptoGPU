
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *y)
{
  // int index = threadIdx.x;
  // int stride = blockDim.x;
  printf("blockIdx.x: %d  threadIdx.x: %d gridDim.x: %d blockDim.x: %d\n", blockDim.x, threadIdx.x, gridDim.x, blockIdx.x);
}

int main(void)
{
  int n = 1;
  float *y;
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&y, n*sizeof(float));
  y[0] = 5;
  add<<<10, 1>>>(n, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)

  // Free memory
  hipFree(y);
  
  return 0;
}