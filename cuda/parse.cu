
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define SIT_SIZE 200
#define NBR_COIN 162
#define BUF 6488
#define MINIMAL_COINS 100

typedef struct {
    double open;
    double high;
    double low;
    double close;
    double volume;
} Data;

typedef struct {
    double time;
    Data data[NBR_COIN];
} Minute;

__global__ void test(Minute **minutes) {
    printf("GLOBAL CALL\n");
    for (int i=0 ;i < 50000 ; i++){
        printf("%lf - %lf\n", minutes[i]->data[3].open, minutes[i]->data[3].volume);
    }
}

int main() {
    // Minute *minute;
    int fd = open("../data/bin/full", O_RDONLY);
    // char *tmpStr = malloc(163);
    // tmpStr[162] = 0;
    // int total = 0;
    Minute **minutes;
    hipMallocManaged(&minutes, sizeof(void **) * 900000);
    int i = -1;
    while (1) {
        i++;
        hipMallocManaged(&minutes[i], sizeof(Minute));
        if (read(fd, minutes[i], BUF) < 1) break;
    }
    test<<<1, 1>>>(minutes);
    hipDeviceSynchronize();
    printf("done\n");
    return 0;
}