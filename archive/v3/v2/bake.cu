#include "hip/hip_runtime.h"
#include "trade.h"

#include "broker.cu"
#include "trade.cu"
#include "analyse.cu"

static __global__ void applyTickBroker(Broker *brokers, int cursor) {
    int workerNbr = threadIdx.x + blockIdx.x * blockDim.x;
    brokers[workerNbr].cursor = cursor;
    tickBroker(&brokers[workerNbr]);
}

// #define TIME_START 700000
#define BAKE_MIN_BETS 200
#define STEP_REG 50000

long long current_timestamp() {
    struct timeval te;
    gettimeofday(&te, NULL);  // get current time
    long long milliseconds =
        te.tv_sec * 1000LL + te.tv_usec / 1000;  // calculate milliseconds
    return milliseconds;
}

static void bake(Data data) {
    int nbrThreads = 128;
    int nbrBlocks = 64;
    int nbrWorkers = nbrThreads * nbrBlocks;
    Broker *brokers;
    hipMallocManaged(&brokers, sizeof(Broker) * nbrWorkers);
    double maxBank = -999999999;
    // double maxReg = 8;
    int totalMinutes = 0;
    long long timeStart = current_timestamp();
    for (int chien = 0; chien < 100000; chien++) {
        for (int i = 0; i < nbrWorkers; i++) {
            brokers[i] = newBroker(data);
        }
        double nbrReg = 0;
        

        for (int i = TIME_START; i < data.nbrMinutes; i++) {
            totalMinutes += 1;
            if (totalMinutes % BROKER_REG_STEP == 0) {
                nbrReg += 1;
                // fprintf(stderr,"perf: %lf\n",
                //        (double)nbrWorkers /
                //            (double)(current_timestamp() - timeStart));
                timeStart = current_timestamp();
            }
            // if (i % 100000 == 0) {
            //     printf("%d / 1300000  wokers : %d\n", i, nbrWorkers);
            // }
            applyTickBroker<<<nbrBlocks, nbrThreads>>>(brokers, i);
        }
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            exit(-1);
        }
        for (int i = 0; i < nbrWorkers; i++) {
            // printf("BK: %-12.2lf\n\n", brokers[i].bank);
            double regScore = brokers[i].reg / nbrReg * 100;
            if (brokers[i].bank > maxBank &&
                regScore > 97 &&
                brokers[i].nbrBets > BAKE_MIN_BETS) {
                // if (brokers[i].bank > 0 && brokers[i].reg >= maxReg &&
                // brokers[i].nbrBets > BAKE_MIN_BETS) {
                printSeed(&brokers[i].seed);
                printf("BK: %-8.02lf FEE: %-8.02lf NB: %-5d REG: %-5.3lf\n\n",
                       brokers[i].bank, brokers[i].fees, brokers[i].nbrBets,
                       regScore);
                maxBank = brokers[i].bank;
                // maxReg = brokers[i].reg;
            }
        }
        // fprintf(stderr, "DONE\n");
    }
}

int main() {
    srand(time(NULL));
    Data data = loadMinutes((char *)"./BTCUSDT");
    bake(data);
    return 0;
}