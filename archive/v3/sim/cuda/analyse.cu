#include "hip/hip_runtime.h"
#include "trade.h"

#define BET_AMOUNT 10000

__device__ double getVariance(Minute *minute, Potards *potards) {
    double min = 99999999999;
    double max = -9999999999;
    for (int cursor = -(potards->period_for_variance + 1); cursor <= 0;
         cursor++) {
        if (minute[cursor].low < min) {
            min = minute[cursor].low;
        }
        if (minute[cursor].high > max) {
            max = minute[cursor].high;
        }
    }
    return (max / min * 100) - 100;
}

__device__ Bet newBet(Minute *minute, int type, double amount, double closeWin,
           double closeLose) {
    Bet bet;
    bet.type = type;
    bet.totalFee = 0;
    bet.amount = amount;
    bet.closeLose = 0;
    bet.closeWin = 0;
    if (bet.type == NO_BET) {
        return bet;
    } else if (bet.type == SELL) {
        bet.closeLose = minute->close * (1 + closeLose * 0.01);
        bet.closeWin = minute->close * (1 - closeWin * 0.01);
    } else if (bet.type == BUY) {
        bet.closeLose = minute->close * (1 - closeLose * 0.01);
        bet.closeWin = minute->close * (1 + closeWin * 0.01);
    }
    return bet;
}

__device__ Bet analyse(Minute *minute, Potards *potards) {
    // dev
    // return newBet(minute, SELL, BET_AMOUNT / minute->close,
    //               potards->closeWin ,
    //               potards->closeLose);

    double change_before_long =
        100 - (minute[-(potards->change_before_long_steps)].close /
               minute->close * 100);
    double variance = getVariance(minute, potards);
    if (change_before_long > potards->change_before_long &&
        variance < potards->maxVariance) {
        return newBet(minute, SELL, BET_AMOUNT / minute->close,
                      potards->closeWin * change_before_long,
                      potards->closeLose * change_before_long);
    } else {
        return newBet(NULL, NO_BET, 0, 0, 0);
    }
}